#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define PI 3.14159265358979323846

__device__ hipDoubleComplex complex_exp(double theta) {
    return make_hipDoubleComplex(cos(theta), sin(theta));
}

__global__ void fft_kernel(hipDoubleComplex *X, int N, int step) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = tid * step * 2;
    if (i + step < N) {
        for (int j = 0; j < step; j++) {
            hipDoubleComplex t = hipCmul(complex_exp(-2.0 * PI * j / (2.0 * step)), X[i + j + step]);
            hipDoubleComplex u = X[i + j];
            X[i + j] = hipCadd(u, t);
            X[i + j + step] = hipCsub(u, t);
        }
    }
}

__global__ void bit_reverse(hipDoubleComplex *X, int N, int logN) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;
    unsigned int rev = 0;
    unsigned int x = tid;
    for (int i = 0; i < logN; i++) {
        rev = (rev << 1) | (x & 1);
        x >>= 1;
    }
    if (rev > tid) {
        hipDoubleComplex temp = X[tid];
        X[tid] = X[rev];
        X[rev] = temp;
    }
}

void cuda_fft(hipDoubleComplex *h_X, int N) {
    hipDoubleComplex *d_X;
    hipMalloc(&d_X, sizeof(hipDoubleComplex) * N);
    hipMemcpy(d_X, h_X, sizeof(hipDoubleComplex) * N, hipMemcpyHostToDevice);
    int logN = log2(N);
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    bit_reverse<<<numBlocks, blockSize>>>(d_X, N, logN);
    hipDeviceSynchronize();
    for (int step = 1; step < N; step *= 2) {
        int numThreads = N / (2 * step);
        int blocks = (numThreads + blockSize - 1) / blockSize;
        fft_kernel<<<blocks, blockSize>>>(d_X, N, step);
        hipDeviceSynchronize();
    }
    hipMemcpy(h_X, d_X, sizeof(hipDoubleComplex) * N, hipMemcpyDeviceToHost);
    hipFree(d_X);
}

int next_power_of_2(int n) {
    int p = 1;
    while (p < n) p <<= 1;
    return p;
}

int main() {
    for (int n = 1; n <= 20; n++) {
        int N = next_power_of_2(n);
        hipDoubleComplex *x = (hipDoubleComplex *)malloc(N * sizeof(hipDoubleComplex));
        for (int i = 0; i < n; i++) {
            double real = rand() % 10;
            double imag = rand() % 10;
            x[i] = make_hipDoubleComplex(real, imag);
        }
        for (int i = n; i < N; i++) {
            x[i] = make_hipDoubleComplex(0, 0);
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        cuda_fft(x, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        printf("n = %2d | Time taken: %.6f ms\n", n, ms);
        free(x);
    }
    return 0;
}
